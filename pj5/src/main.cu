#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "render.h"

using namespace std;

#define MAX_BLOCKS_PER_DIM 65535


void write(string filename, double* G, int len){
	ofstream of(filename,
		ios::trunc | ios::in | ios::out | ios::binary);
	of.write(reinterpret_cast<char *>(G), len*sizeof(double));
	of.close();
}


void run(int argc, char** argv){
	//read common args
	string mode = string{argv[1]};
	int ndim = atoi(argv[2]);
	if(mode != "serial" && mode != "cuda"){
		cout << "Unrecognized mode!" << endl;
		return ;
	}
	
	//int blocks, threads_per_block;
	//blocks = MIN(rays/threads_per_block + 1, MAX_BLOCKS_PER_DIM);

	//rendering...
	cout << "Running " << mode << " Ray Tracing..." << endl;

	Scene p({10,10,10}, {0,12,0}, {4,4,-1}, 6);
	double *G = (double*)malloc(sizeof(double)*ndim*ndim);
	for(auto i=0; i<ndim*ndim; ++i)
		G[i] = 0;

	hipEvent_t start_device, stop_device;
	float time_device;
	hipEventCreate(&start_device);
	hipEventCreate(&stop_device);

	hipEventRecord(start_device, 0);

	if(mode == "serial"){
		int rays = atoi(argv[3]);
  		render_serial(p, ndim, G, rays);
	}
	else if(mode == "cuda"){
		int b = atoi(argv[3]),
			th = atoi(argv[4]);

		Scene *dev_p;
		double *dev_G;
		hipMalloc( (void **) &dev_p, sizeof(Scene));
		hipMalloc( (void **) &dev_G, sizeof(double)*ndim*ndim);
		hipMemcpy(dev_p, &p, sizeof(Scene), hipMemcpyHostToDevice);
		hipMemcpy(dev_G, G, sizeof(double)*ndim*ndim, hipMemcpyHostToDevice);

	  	render_cuda<<<b, th>>>(dev_p, ndim, dev_G);

		hipMemcpy(G, dev_G, sizeof(double)*ndim*ndim, hipMemcpyDeviceToHost);
		hipFree(dev_p); 
		hipFree(dev_G); 
	}

	hipEventRecord(stop_device, 0);
	hipEventSynchronize(stop_device);
	hipEventElapsedTime(&time_device, start_device, stop_device);
	cout << "Running time: "
		 << time_device << " ms" << endl;

	hipEventDestroy(start_device);
	hipEventDestroy(stop_device);

	//write file
	write(mode+".out", G, ndim*ndim);

	free(G);
}


int main(int argc, char **argv){

	run(argc, argv);

	return 0;
}